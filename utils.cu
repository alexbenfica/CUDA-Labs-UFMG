// Fonctions auxiliaires
#ifdef _WIN32
#define WIN32_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#else

#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <time.h>
#endif

/**
 * Generate Random number in single precision.
 */
float random_float(int emin, int emax, int pos_neg){
    double tmp;
    unsigned int i, val;
    int e;

    val = (rand() & 0x000000ff);
    for(i=0; i<(sizeof(int)); i++){
	val = val << 8;
	val += (rand() & 0x000000ff ); /* we keep only 8 bits */
    }
    e = emin + (int)( (double)rand()*(emax-emin)/(double)RAND_MAX);
    tmp = ldexp(1.0 + (double)val / UINT_MAX, e);
    if ((pos_neg) && (rand() > (RAND_MAX/2)))		tmp *= -1;

    return (float)tmp;
}

double getclock()
{
#ifdef _WIN32
    LARGE_INTEGER li;
    QueryPerformanceFrequency(&li);

    double PCFreq = (double)li.QuadPart;
    QueryPerformanceCounter(&li);
    __int64 timerStart = li.QuadPart;
    return ((double)li.QuadPart)/PCFreq;
#else
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec + double(tv.tv_usec) / 1000000.;
#endif
}









/**
 * Returns the real time, in seconds, or -1.0 if an error occurred.
 *
 * Time is measured since an arbitrary and OS-dependent start time.
 * The returned real time is only useful for computing an elapsed time
 * between two calls to this function.
 */
double getRealTime( )
{
#if defined(_WIN32)
	FILETIME tm;
	ULONGLONG t;
#if defined(NTDDI_WIN8) && NTDDI_VERSION >= NTDDI_WIN8
	/* Windows 8, Windows Server 2012 and later. ---------------- */
	GetSystemTimePreciseAsFileTime( &tm );
#else
	/* Windows 2000 and later. ---------------------------------- */
	GetSystemTimeAsFileTime( &tm );
#endif
	t = ((ULONGLONG)tm.dwHighDateTime << 32) | (ULONGLONG)tm.dwLowDateTime;
	return (double)t / 10000000.0;

#elif (defined(__hpux) || defined(hpux)) || ((defined(__sun__) || defined(__sun) || defined(sun)) && (defined(__SVR4) || defined(__svr4__)))
	/* HP-UX, Solaris. ------------------------------------------ */
	return (double)gethrtime( ) / 1000000000.0;

#elif defined(__MACH__) && defined(__APPLE__)
	/* OSX. ----------------------------------------------------- */
	static double timeConvert = 0.0;
	if ( timeConvert == 0.0 )
	{
		mach_timebase_info_data_t timeBase;
		(void)mach_timebase_info( &timeBase );
		timeConvert = (double)timeBase.numer /
			(double)timeBase.denom /
			1000000000.0;
	}
	return (double)mach_absolute_time( ) * timeConvert;

#elif defined(_POSIX_VERSION)
	/* POSIX. --------------------------------------------------- */
#if defined(_POSIX_TIMERS) && (_POSIX_TIMERS > 0)
	{
		struct timespec ts;
#if defined(CLOCK_MONOTONIC_PRECISE)
		/* BSD. --------------------------------------------- */
		const clockid_t id = CLOCK_MONOTONIC_PRECISE;
#elif defined(CLOCK_MONOTONIC_RAW)
		/* Linux. ------------------------------------------- */
		const clockid_t id = CLOCK_MONOTONIC_RAW;
#elif defined(CLOCK_HIGHRES)
		/* Solaris. ----------------------------------------- */
		const clockid_t id = CLOCK_HIGHRES;
#elif defined(CLOCK_MONOTONIC)
		/* AIX, BSD, Linux, POSIX, Solaris. ----------------- */
		const clockid_t id = CLOCK_MONOTONIC;
#elif defined(CLOCK_REALTIME)
		/* AIX, BSD, HP-UX, Linux, POSIX. ------------------- */
		const clockid_t id = CLOCK_REALTIME;
#else
		const clockid_t id = (clockid_t)-1;	/* Unknown. */
#endif /* CLOCK_* */
		if ( id != (clockid_t)-1 && clock_gettime( id, &ts ) != -1 )
			return (double)ts.tv_sec +
				(double)ts.tv_nsec / 1000000000.0;
		/* Fall thru. */
	}
#endif /* _POSIX_TIMERS */

	/* AIX, BSD, Cygwin, HP-UX, Linux, OSX, POSIX, Solaris. ----- */
	struct timeval tm;
	gettimeofday( &tm, NULL );
	return (double)tm.tv_sec + (double)tm.tv_usec / 1000000.0;
#else
	return -1.0;		/* Failed. */
#endif
}