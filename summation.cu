#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdlib.h>
#include <time.h>

struct results
{
	float sum;
};

#include "summation_kernel.cu"

// CPU implementation
float log2_series(int n)
{    
    int i = 0;
    double sum = 0;
    for(i=n-1;i>=0;i--){
        sum += ((((i%2)-1.0) + (i%2)) * (-1.0)) / ((double)i + 1.0);
        //printf("%.9f\n",sum);
    }
    //printf("%.50f\n", sum);
    //printf("%.50e\n", sum - sum1);
    return sum;
}







int main(int argc, char ** argv)
{
    
    if(argc < 4){                
        printf("\nYou must specify: kernel_id,  number of blocks, and number of threads per block.\n");
        return 1;
    }
    
    int data_size = 1024 * 1024 * 128;

    
    // Run CPU version
    clock_t start_cpu = clock();    
    float log2 = log2_series(data_size);                    
    clock_t end_cpu = clock();    
    float seconds = (float)(end_cpu - start_cpu) / CLOCKS_PER_SEC;    
    printf("\nlog(2)    = %20.20f", log(2.0));    
    printf("\nCPU RESULT: %20.20f\n", log2);        
    printf(" Total time :%fs\n", seconds);



    
    // Parameter definition (original from example...)
    int blocks_in_grid = 8;    
    int threads_per_block = 4 * 32;

    // Modified parameters for testing purposes...
    // Some ideias about how to setup the block an thread number
    // http://stackoverflow.com/questions/4861244/how-many-threads-does-nvidia-gts-450-has
    blocks_in_grid = atoi(argv[2]);
    threads_per_block = atoi(argv[3]);

    
    int num_threads = threads_per_block * blocks_in_grid;
    

    // Timer initialization and configuration
    hipEvent_t start, stop;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));

    // Each thread will returno only one element.
    int results_size = num_threads;
    
    // data_out_cpu is a pointer of type results
    results* data_out_cpu;
    results* data_out_gpu;
    
    // Allocating output data on CPU
    // Cast necessary to ensure corret type on data_out_cpu
    data_out_cpu = (results *) malloc(sizeof(results) * results_size);

    // Allocating output data on GPU    
    hipMalloc((void**)&data_out_gpu, sizeof(results) * results_size);

    // Start timer
    CUDA_SAFE_CALL(hipEventRecord(start, 0));

    
    
    // Executes kernel, depending on input parameters...        
    int kernel_id = atoi(argv[1]);        
    
    switch(kernel_id){
        case 0:
            summation_kernel_0<<<blocks_in_grid, threads_per_block>>>(data_size, data_out_gpu);
            break;
        case 1:
            summation_kernel_1<<<blocks_in_grid, threads_per_block>>>(data_size, data_out_gpu);
            break;
    }
    
    
    // Stop timer
    CUDA_SAFE_CALL(hipEventRecord(stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop));
    
    // Get results back from GPU to CPU memory
    hipMemcpy(data_out_cpu, data_out_gpu, sizeof(results) * results_size, hipMemcpyDeviceToHost);
    
    // Finish reduction on CPU, adding all elements
    int i;
    float sum = 0.;
    printf("\n");
    for(i=0; i<num_threads; i++){
        sum += data_out_cpu[i].sum;        
        #if 0
        if((i>0)&&(i<40)){
            printf("Thread %d result: %20.20f\n" , i, data_out_cpu[i].sum);
        }
        #endif  
    }
    
    
    
    
    // Cleanup CPU and GPU memory.
    hipFree(data_out_gpu);
    free(data_out_cpu);
    
    
    
    // Show timming statistics
    
    printf("\nlog(2)   = %20.20f", log(2.0));
    
    printf("\nGPU RESULT:%20.20f\n", sum);
    printf(" Kernel ID: %d\n", kernel_id);
    printf(" Blocks: %d\n", blocks_in_grid);
    printf(" Thread per block: %d\n", threads_per_block);
    printf(" Total threads: %d\n", threads_per_block * blocks_in_grid);
    
    float elapsedTime;
    // In ms
    CUDA_SAFE_CALL(hipEventElapsedTime(&elapsedTime, start, stop));	
    CUDA_SAFE_CALL(hipEventDestroy(start));
    CUDA_SAFE_CALL(hipEventDestroy(stop));
    
    double total_time = elapsedTime / 1000.;	// s
    double time_per_iter = total_time / (double)data_size;
    double bandwidth = sizeof(float) / time_per_iter; // B/s
    
    printf(" Per iteration: %g ns\n Throughput: %g GB/s\n Total time: %gs\n",    	
    	time_per_iter * 1.e9,
    	bandwidth / 1.e9,
        total_time);
  
    printf("\n Speedup CPU to GPU: %5.2fx" , ((double)seconds / total_time));
    
    return 0;
}

