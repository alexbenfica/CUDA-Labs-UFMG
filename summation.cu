#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
#include "summation_kernel.cu"




// CPU implementation
float log2_series(int n){    
    int i = 0;
    double sum = 0;
    for(i=n-1;i>=0;i--){
        sum += ((((i%2)-1.0) + (i%2)) * (-1.0)) / ((double)i + 1.0);
        //printf("%.9f\n",sum);
    }
    //printf("%.50f\n", sum);
    //printf("%.50e\n", sum - sum1);
    return sum;
}







int main(int argc, char ** argv)
{
    
    if(argc < 4){                
        printf("\nYou must specify: kernel_id,  number of blocks, and number of threads per block.\n");
        return 1;
    }
    
    // Executes kernel, depending on input parameters...        
    int kernel_id = atoi(argv[1]);        
    
    
    int data_size = 1024 * 1024 * 128;

    
    // Run CPU version
    clock_t start_cpu = clock();    
    float log2 = log2_series(data_size);                    
    clock_t end_cpu = clock();    
    float seconds = (float)(end_cpu - start_cpu) / CLOCKS_PER_SEC;    
    printf("\nlog(2)    = %20.20f", log(2.0));    
    printf("\nCPU RESULT: %20.20f\n", log2);        
    printf(" Total time :%fs\n", seconds);

    // Parameter definition (original from example...)
    int blocks_in_grid = 8;    
    int threads_per_block = 4 * 32;

    // Modified parameters for testing purposes...
    // Some ideias about how to setup the block an thread number
    // http://stackoverflow.com/questions/4861244/how-many-threads-does-nvidia-gts-450-has
    blocks_in_grid = atoi(argv[2]);
    threads_per_block = atoi(argv[3]);

    
    int num_threads = threads_per_block * blocks_in_grid;
    

    // Timer initialization and configuration
    hipEvent_t start, stop;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));

    
    int results_size;
    
    switch(kernel_id){
        case SUM:            
        case SUM_INTERLEAVED:
            // Each thread will return only one element as a result.
            results_size = num_threads;            
            break;
        
        case SUM_BLOCK:
        case SUM_GPU_ONLY:
            // Only one element will be returned per block
            results_size = blocks_in_grid;            
            break;                       
            
    }

    
    // data_out_cpu is a pointer of type results
    results* data_out_cpu;
    results* data_out_gpu;
    results* data_out_reduce;
    
    // Allocating output data on CPU
    // Cast necessary to ensure corret type on data_out_cpu
    data_out_cpu = (results *) malloc(sizeof(results) * results_size);

    // Allocating output data on GPU    
    hipMalloc((void**)&data_out_gpu, sizeof(results) * results_size);

    
    if(kernel_id == SUM_GPU_ONLY){        
        // Allocating output data for reduce on GPU    
        hipMalloc((void**)&data_out_reduce, sizeof(results) * results_size);        
    }
    
    
    // Start timer
    CUDA_SAFE_CALL(hipEventRecord(start, 0));

    
    switch(kernel_id){
        case SUM:
            summation_kernel_0<<<blocks_in_grid, threads_per_block>>>(data_size, data_out_gpu);
            break;
        case SUM_INTERLEAVED:
            summation_kernel_interleaved<<<blocks_in_grid, threads_per_block>>>(data_size, data_out_gpu);
            break;
        case SUM_BLOCK:            
            summation_kernel_value_per_block<<<blocks_in_grid, threads_per_block, threads_per_block*sizeof(float)>>>(data_size, data_out_gpu);
            break;
        case SUM_GPU_ONLY:            
            summation_kernel_gpu_only<<<blocks_in_grid, threads_per_block, threads_per_block*sizeof(float)>>>(data_size, data_out_gpu);
            reduce<<<blocks_in_grid, threads_per_block, threads_per_block*sizeof(float)>>>(blocks_in_grid, data_out_gpu, data_out_reduce);
            break;                        
    }
    
    
    // Stop timer
    CUDA_SAFE_CALL(hipEventRecord(stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop));

    switch(kernel_id){
        case SUM:
        case SUM_INTERLEAVED:
        case SUM_BLOCK:            
            // Get results back from GPU to CPU memory
            hipMemcpy(data_out_cpu, data_out_gpu, sizeof(results) * results_size, hipMemcpyDeviceToHost);
            break;
        case SUM_GPU_ONLY:
            // Copy only one element back from GPU to CPU
            hipMemcpy(data_out_cpu, data_out_reduce, sizeof(results) * 1, hipMemcpyDeviceToHost);
            break;            
    }

    
    
    
    int i;
    float sum = 0.;    
    switch(kernel_id){
        // Finish reduction on CPU, adding all elements
        case SUM:
        case SUM_INTERLEAVED:
        case SUM_BLOCK:            
            printf("\n");
            for(i=0; i<results_size; i++){
                sum += data_out_cpu[i].sum;        
                #if 0
                if((i>0)&&(i<40)){
                    printf("Thread %d result: %20.20f\n" , i, data_out_cpu[i].sum);
                }
                #endif  
            }
            break;
        case SUM_GPU_ONLY:
            sum = data_out_cpu[0].sum;        
            break;
    }
            
    
    // Cleanup CPU and GPU memory.
    hipFree(data_out_gpu);
    free(data_out_cpu);
    
    
    
    // Show timming statistics
    
    printf("\nlog(2)   = %20.20f", log(2.0));
    
    printf("\nGPU RESULT:%20.20f\n", sum);
    printf(" Kernel ID: %d\n", kernel_id);
    printf(" Blocks: %d\n", blocks_in_grid);
    printf(" Thread per block: %d\n", threads_per_block);
    printf(" Total threads: %d\n", threads_per_block * blocks_in_grid);
    
    float elapsedTime;
    // In ms
    CUDA_SAFE_CALL(hipEventElapsedTime(&elapsedTime, start, stop));	
    CUDA_SAFE_CALL(hipEventDestroy(start));
    CUDA_SAFE_CALL(hipEventDestroy(stop));
    
    double total_time = elapsedTime / 1000.;	// s
    double time_per_iter = total_time / (double)data_size;
    double bandwidth = sizeof(float) / time_per_iter; // B/s
    
    printf(" Per iteration: %g ns\n Throughput: %g GB/s\n Total time: %gs\n",    	
    	time_per_iter * 1.e9,
    	bandwidth / 1.e9,
        total_time);
  
    printf("\n Speedup CPU to GPU: %5.2fx" , ((double)seconds / total_time));
    
    return 0;
}

