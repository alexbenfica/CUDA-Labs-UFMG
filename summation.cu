#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdlib.h>

struct results
{
	float sum;
};

#include "summation_kernel.cu"

// CPU implementation
float log2_series(int n)
{    
    int i = 0;
    double sum = 0;
    for(i=n-1;i>=0;i--){
        sum += ((((i%2)-1.0) + (i%2)) * (-1.0)) / ((double)i + 1.0);
        //printf("%.9f\n",sum);
    }
    //printf("%.50f\n", sum);
    //printf("%.50e\n", sum - sum1);
    return sum;
}







int main(int argc, char ** argv)
{
    int data_size = 1024 * 1024 * 128;

    // Run CPU version
    double start_time = 0;
    //double start_time = getclock();
    float log2 = log2_series(data_size);
    double end_time = 1;
    //double end_time = getclock();
    
    printf("CPU result: %f\n", log2);
    printf(" log(2)=%f\n", log(2.0));
    printf(" time=%fs\n", end_time - start_time);

    
    // Parameter definition
    int threads_per_block = 4 * 32;
    int blocks_in_grid = 8;    
    int num_threads = threads_per_block * blocks_in_grid;

    // Timer initialization and configuration
    hipEvent_t start, stop;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));

    // Each thread will returno only one element.
    int results_size = num_threads;
    
    // data_out_cpu is a pointer of type results
    results* data_out_cpu;
    results* data_out_gpu;
    
    // Allocating output data on CPU
    // Cast necessary to ensure corret type on data_out_cpu
    data_out_cpu = (results *) malloc(sizeof(results) * results_size);

    // Allocating output data on GPU    
    printf("%d", hipMalloc((void**)&data_out_gpu, sizeof(results) * results_size));

    // Start timer
    CUDA_SAFE_CALL(hipEventRecord(start, 0));

    
    
    // Execute kernel
    summation_kernel<<<1, num_threads>>>(data_size / num_threads, data_out_gpu);
    
    
    
    // Stop timer
    CUDA_SAFE_CALL(hipEventRecord(stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop));
    
    // Get results back from GPU to CPU memory
    hipMemcpy(data_out_cpu, data_out_gpu, sizeof(results) * results_size, hipMemcpyDeviceToHost);
    
    // Finish reduction on CPU, adding all elements
    int i;
    float sum = 0.;
    for(i=0; i<num_threads; i++){
        sum += data_out_cpu[i].sum;        
    }
    
    
    // Cleanup CPU and GPU memory.
    hipFree(data_out_gpu);
    free(data_out_cpu);
    
    
    
    // Show timming statistics
    
    printf("GPU results:\n");
    printf(" Sum: %f\n", sum);
    
    
    float elapsedTime;
    // In ms
    CUDA_SAFE_CALL(hipEventElapsedTime(&elapsedTime, start, stop));	
    CUDA_SAFE_CALL(hipEventDestroy(start));
    CUDA_SAFE_CALL(hipEventDestroy(stop));
    
    double total_time = elapsedTime / 1000.;	// s
    double time_per_iter = total_time / (double)data_size;
    double bandwidth = sizeof(float) / time_per_iter; // B/s
    
    printf(" Total time: %g s,\n Per iteration: %g ns\n Throughput: %g GB/s\n",
    	total_time,
    	time_per_iter * 1.e9,
    	bandwidth / 1.e9);
  
    return 0;
}

